// C++ program for Merge Sort

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// Merges two subarrays of array[].
// First subarray is arr[begin..mid]
// Second subarray is arr[mid+1..end]
void merge(int array[], int *leftArray, int *rightArray, int subArrayOne, int subArrayTwo)
{

    auto indexOfSubArrayOne = 0, indexOfSubArrayTwo = 0;
    int indexOfMergedArray = 0;

    // Merge the temp arrays back into array[left..right]
    while (indexOfSubArrayOne < subArrayOne && indexOfSubArrayTwo < subArrayTwo)
    {
        if (leftArray[indexOfSubArrayOne] <= rightArray[indexOfSubArrayTwo])
        {
            array[indexOfMergedArray] = leftArray[indexOfSubArrayOne];
            indexOfSubArrayOne++;
        }
        else
        {
            array[indexOfMergedArray] = rightArray[indexOfSubArrayTwo];
            indexOfSubArrayTwo++;
        }
        indexOfMergedArray++;
    }

    // Copy the remaining elements of
    // left[], if there are any
    while (indexOfSubArrayOne < subArrayOne)
    {
        array[indexOfMergedArray] = leftArray[indexOfSubArrayOne];
        indexOfSubArrayOne++;
        indexOfMergedArray++;
    }

    // Copy the remaining elements of
    // right[], if there are any
    while (indexOfSubArrayTwo < subArrayTwo)
    {
        array[indexOfMergedArray] = rightArray[indexOfSubArrayTwo];
        indexOfSubArrayTwo++;
        indexOfMergedArray++;
    }
    delete[] leftArray;
    delete[] rightArray;
}

// begin is for left index and end is right index
// of the sub-array of arr to be sorted
void mergeSort(int array[], int const begin, int const end)
{
    if (begin >= end)
        return;

    int mid = begin + (end - begin) / 2;
    int arraySize = (end - begin + 1);
    mergeSort(array, begin, mid);
    mergeSort(array, mid + 1, end);
    int const subArrayOne = mid - begin + 1;
    int const subArrayTwo = end - mid;

    // Create temp arrays
    auto *leftArray = new int[subArrayOne],
         *rightArray = new int[subArrayTwo];

    // Copy data to temp arrays leftArray[] and rightArray[]
    for (auto i = 0; i < subArrayOne; i++)
        leftArray[i] = array[begin + i];
    for (auto j = 0; j < subArrayTwo; j++)
        rightArray[j] = array[mid + 1 + j];

    merge(&array[begin], leftArray, rightArray, subArrayOne, subArrayTwo);
    cout << "Merging: " << begin << " " << mid << " " << end << endl;
    for (int i = 0; i < arraySize; i++)
    {
        cout << array[i + begin] << " ";
    }
    cout << endl;
}

// UTILITY FUNCTIONS
// Function to print an array
void printArray(int A[], int size)
{
    for (int i = 0; i < size; i++)
        cout << A[i] << " ";
    cout << endl;
}

// Driver code
int main()
{
    int n = 2500;
    // take n as input
    cout << "Enter the size of the array: ";
    cin >> n;
    int *a = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++)
    {
        a[i] = rand() % (n / 2);
    }
    // printArray(a, n);
    cout << "Size: " << n << endl;
    // cout << "Given array is \n";
    // printArray(a, n);

    mergeSort(a, 0, n - 1);

    cout << "\nSorted array is \n";
    printArray(a, n);
}

// This code is contributed by Mayank Tyagi
// This code was revised by Joshua Estes
